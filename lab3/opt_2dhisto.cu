#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

#define BIN_COUNT 1024

__global__ void generate_hist(uint32_t* input, uint32_t* global_bins)
{
	int Tid  = blockIdx.x * blockDim.x + threadIdx.x;
	int numThreads = blockDim.x * gridDim.x;
	
    // shared memory to store partial histogram data
	__shared__ int s_Hist[BIN_COUNT];	

	// Clear the buffer before using
	for (int pos = threadIdx.x; pos < BIN_COUNT; pos += blockDim.x) {
		s_Hist[pos] = 0;
	}
	__syncthreads ();

	// Start calculating partial Histogram
	for (int pos = Tid; pos < INPUT_HEIGHT * INPUT_WIDTH; pos += numThreads) {
		//if (s_Hist[input[pos]] < 255) {
			atomicAdd(s_Hist + input[pos], 1);
		//}
	}
	__syncthreads();

	//update global histogram
	for(int pos = threadIdx.x; pos < BIN_COUNT; pos += numThreads) {
		//if(global_bins[threadIdx.x] < 255) {
			atomicAdd(global_bins + pos, s_Hist[pos]);
		//}
	}
}

__global__ void convertTo8(uint32_t* global_bins, uint8_t* device_bins)
{
	int Tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(global_bins[Tid] < 255) {
		device_bins[Tid] = (uint8_t)global_bins[Tid];
	}
	else {
		device_bins[Tid] = (uint8_t)255;
	}	
}

void* AllocateDevice(size_t size)
{
	void *addr;
	hipMalloc(&addr, size);
	return addr;
}

void MemCpyToDevice(void* dest, void* src, size_t size)
{
	hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

void CopyFromDevice(void* dest, void* src, size_t size)
{
	hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
}

void FreeDevice(void* addr)
{
	hipFree(addr);
}

void opt_2dhisto(uint32_t* device_input, uint32_t* global_bins, uint8_t* device_bins)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */
    hipMemset(global_bins, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));
    generate_hist<<<16, 1024>>>(device_input, global_bins);
	convertTo8<<<1, 1024>>>(global_bins, device_bins);
	hipDeviceSynchronize();
}

